#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""
#include "math_functions.h"

#include "FSDAF.h"

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}


__host__ __device__
void sortSimilarPixel(float arr[], size_t id[], size_t len, size_t num_cand)
{
	for (size_t i = 0; i < num_cand; i++)
	{
		float fMaxArr = arr[i];
		size_t nID = id[i];
		size_t _iCount = i;
		for (size_t j = i + 1; j < len; j++)
		{
			if (arr[j] < fMaxArr)
			{
				fMaxArr = arr[j];
				nID = id[j];
				_iCount = j;
			}
		}
		arr[_iCount] = arr[i];
		id[_iCount] = id[i];
		arr[i] = fMaxArr;
		id[i] = nID;
	}
}


int getBlockLine(parameter* p1, int &nL, int ns, int nb, int num_similar_pixel)
{
	size_t a = 0;

	hipDeviceProp_t deviceProp;
	int deviceCount;
	CHECK(hipGetDeviceCount(&deviceCount));
	for (size_t i = 0; i < deviceCount; i++)
	{
		CHECK(hipGetDeviceProperties(&deviceProp, i));
		a = deviceProp.totalGlobalMem;
	}

	int nThread = p1->dimGrid.x * p1->dimBlock.x;			//size of 1 cycle
	int nWindowSize = (p1->w + 1 + p1->w) * (p1->w + 1 + p1->w);	//size of window, and w is the length of the searching window

	//Constant memory of final computing
	int nConstantMemory = sizeof(float) * nWindowSize + sizeof(int) * (p1->w + 1) * (p1->w + 1) * nWindowSize * 2
		+ sizeof(int) * nWindowSize * nThread * 2 + sizeof(float) * nWindowSize * nThread + sizeof(double) * num_similar_pixel * nThread;

	//Memory of each line
	int nSizeOfEachLine = (sizeof(float) * 3 + sizeof(short) * 3) * ns * nb;

	nL = (a * 0.8 - nConstantMemory) / nSizeOfEachLine;

	return 0;
}

__global__
void cuHet(int iCycle, int nThread, int scale_d, int ns_block, int nl_block, short* L1_class_d, float* het_index_d)
{
	size_t iThread = blockIdx.x * blockDim.x + threadIdx.x;

	if (iThread < nThread)
	{
		if (iThread + iCycle * nThread < ns_block * nl_block)
		{
			size_t i = (iThread + iCycle * nThread) % ns_block;
			size_t j = (iThread + iCycle * nThread) / ns_block;

			size_t ai = 0;
			size_t bi = 0;
			size_t aj = 0;
			size_t bj = 0;

			if (i < scale_d) ai = 0; else ai = i - scale_d;
			if (i + scale_d > ns_block - 1) bi = ns_block - 1; else bi = i + scale_d;
			if (j < scale_d) aj = 0; else aj = j - scale_d;
			if (j + scale_d > nl_block - 1) bj = nl_block - 1; else bj = j + scale_d;

			size_t num_sameclass = 0;
			for (size_t l = aj; l < bj; l++)
				for (size_t k = ai; k < bi; k++)
					if (L1_class_d[k + l * ns_block] == L1_class_d[i + j * ns_block])
						num_sameclass++;
			het_index_d[i + j * ns_block] = float(num_sameclass) / ((bi - ai)*(bj - aj));

		}
	}
}

int cuGetHetIndex(parameter* p1, int scale_d, int ns_block, int nl_block, short* L1_class, float* het_index)
{
	dim3 dimGrid = p1->dimGrid;
	dim3 dimBlock = p1->dimBlock;

	size_t nThread = dimGrid.x * dimBlock.x;			//size of 1 cycle

	// Malloc the memory in device(GPU)
	short *L1_class_d;
	float *het_index_d;

	hipMalloc((void**)&L1_class_d, sizeof(short) * ns_block * nl_block);
	hipMalloc((void**)&het_index_d, sizeof(float) * ns_block * nl_block);

	// Copy data from host to device (CPU to GPU)
	hipMemcpy(L1_class_d, L1_class, sizeof(short) * ns_block * nl_block, hipMemcpyHostToDevice);
	hipMemcpy(het_index_d, het_index, sizeof(float) * ns_block * nl_block, hipMemcpyHostToDevice);

	for (int iCycle = 0; iCycle * dimGrid.x * dimBlock.x < ns_block * nl_block; iCycle++)
	{
		cuHet << <dimGrid, dimBlock >> > (iCycle, nThread, scale_d, ns_block, nl_block, L1_class_d, het_index_d);
		CHECK(hipDeviceSynchronize());
	}

	hipMemcpy(het_index, het_index_d, sizeof(float) *  ns_block * nl_block, hipMemcpyDeviceToHost);

	hipFree(L1_class_d);
	hipFree(het_index_d);

	CHECK(hipDeviceReset());

	return 0;
}

__global__
void cuIDW(int icycle, int nThread, int scale_factor, int background_band, int ns_block, int nl_block, int nb, int ns_c, int nl_c, int IDWSearchRadius, int IDWPower, int *col_c_D, int *row_c_D, float *coarse_c2_D, float* L2_IDW_D)
{
	size_t iThread = blockIdx.x * blockDim.x + threadIdx.x;

	if (iThread < nThread)
	{
		size_t i = iThread + icycle * nThread;

		if (i < ns_block * nl_block)
		{
			for (size_t ib = 0; ib < nb; ib++)
			{
				double wSum = 0;	// weight sum

				//coordinate indexes of the minimum enclosing rectangle for the searching circle 
				int index0 = (i % ns_block + scale_factor / 2);
				int index1 = (i % ns_block + IDWSearchRadius - (scale_factor - 1) / 2) / scale_factor;
				int index2 = (i / ns_block + scale_factor / 2);
				int index3 = (i / ns_block + IDWSearchRadius - (scale_factor - 1) / 2) / scale_factor;

				if (index0 < IDWSearchRadius) index0 = 0; else index0 = (index0 - IDWSearchRadius) / scale_factor;
				if (index1 > ns_c - 1) index1 = ns_c - 1;
				if (index2 < IDWSearchRadius) index2 = 0; else index2 = (index2 - IDWSearchRadius) / scale_factor;
				if (index3 > nl_c - 1) index3 = nl_c - 1;

				for (size_t _j = index2; _j <= index3; _j++)
				{
					if (abs(wSum - 1) <= 1.0e-8)
						break;
					for (size_t _i = index0; _i <= index1; _i++)
					{
						size_t iKnownPoint = _i + _j * ns_c;
						double d = sqrt(pow((double)(col_c_D[iKnownPoint] - (int(i) % ns_block)), 2) + pow((double)(row_c_D[iKnownPoint] - (int(i) / ns_block)), 2));
						
						if (abs(d) >= 1.0e-8)		//if sqrt((knownPoint[nKnownPointIndex].x - (i % ns_block))^2 + (y - yi)^2 ) <= IDWSearchRadius, ...
						{
							if (d <= IDWSearchRadius)
							{
								wSum += pow(d, -IDWPower);
								L2_IDW_D[i + ib * ns_block * nl_block] += pow(d, -IDWPower) * coarse_c2_D[iKnownPoint + ib * ns_c * nl_c];
							}
						}
						else
						{
							L2_IDW_D[i + ib * ns_block * nl_block] = coarse_c2_D[iKnownPoint + ib * ns_c * nl_c];
							wSum = 1;
							break;
						}
					}
				}

				if (abs(wSum) > 1.0e-8)
					L2_IDW_D[i + ib * ns_block * nl_block] /= wSum;
				else
				{
					L2_IDW_D[i + ib * ns_block * nl_block] = 0;
					printf("IDW=0");
				}
			}
		}
	}
}

int cuInterpolate_IDW(parameter* p1, int ns_block, int nl_block, int nb, int ns_c, int nl_c, int *col_c, int *row_c, float *coarse_c2, float* L2_IDW)
{
	dim3 dimGrid = p1->dimGrid;
	dim3 dimBlock = p1->dimBlock;
	size_t nThread = dimGrid.x * dimBlock.x;			//size of 1 cycle

	// Malloc the memory in device(GPU)
	int *col_c_d, *row_c_d;
	float *coarse_c2_d, *L2_IDW_d;
	hipMalloc((void**)&col_c_d, sizeof(int) * ns_c * nl_c);
	hipMalloc((void**)&row_c_d, sizeof(int) * ns_c * nl_c);
	hipMalloc((void**)&coarse_c2_d, sizeof(float) * ns_c * nl_c * nb);
	hipMalloc((void**)&L2_IDW_d, sizeof(float) * ns_block * nl_block * nb);

	// Copy data from host to device (CPU to GPU)
	hipMemcpy(col_c_d, col_c, sizeof(int) * ns_c * nl_c, hipMemcpyHostToDevice);
	hipMemcpy(row_c_d, row_c, sizeof(int) * ns_c * nl_c, hipMemcpyHostToDevice);
	hipMemcpy(coarse_c2_d, coarse_c2, sizeof(float) * ns_c * nl_c * nb, hipMemcpyHostToDevice);
	hipMemcpy(L2_IDW_d, L2_IDW, sizeof(float) * ns_block * nl_block * nb, hipMemcpyHostToDevice);
	
	for (size_t iCycle = 0; iCycle * dimGrid.x * dimBlock.x < ns_block * nl_block; iCycle++)
	{
		cuIDW << <dimGrid, dimBlock >> > (iCycle, nThread, p1->scale_factor, p1->background_band, ns_block, nl_block, nb, ns_c, nl_c, p1->IDWSearchRadius, p1->IDWPower, col_c_d, row_c_d, coarse_c2_d, L2_IDW_d);
		CHECK(hipDeviceSynchronize());
	}

	hipMemcpy(L2_IDW, L2_IDW_d, sizeof(float) *  ns_block * nl_block * nb, hipMemcpyDeviceToHost);

	hipFree(col_c_d);
	hipFree(row_c_d);
	hipFree(coarse_c2_d);
	hipFree(L2_IDW_d);

	CHECK(hipDeviceReset());

	return 0;
}

__global__
void cuFine2_1(float* FineImg1_d, float *CoarseImg1_d, float *CoarseImg2_d, float *change_21_d, float *fine2_d,
	float *D_D_all_d, float *similar_th_d, size_t *col_wind_d, size_t *row_wind_d, size_t *positionCand_orderDis_d, float *mmap_order_dis_d, double *D_D_cand_d, size_t ns, size_t nl, int nb, int background_band,
	int background, size_t w, float DN_max, float DN_min, size_t num_similar_pixel, size_t cycle_size, size_t cycle_time)
{
	size_t i_thread = blockIdx.x * blockDim.x + threadIdx.x;

	if (i_thread < cycle_size)
	{
		size_t i = (i_thread + cycle_size * cycle_time) % ns;
		size_t j = (i_thread + cycle_size * cycle_time) / ns;
		if (i < ns && j < nl &&
			(CoarseImg1_d[i + j * ns + (background_band - 1) * ns * nl] - background) > 1e-6)
		{
			// searching range
			size_t ai = max(i, w) - w;
			size_t bi = min(ns - 1, i + w);
			size_t aj = max(j, w) - w;
			size_t bj = min(nl - 1, j + w);
			size_t ci = i - ai;
			size_t cj = j - aj;
			size_t nI = w, nJ = w;
			if (i - w < 0) nI = i;
			if (j - w < 0) nJ = j;
			if (i + w > ns - 1) nI = ns - 1 - i;
			if (j + w > nl - 1) nJ = nl - 1 - j;
		
			//search similar pixels within the window
			size_t nWindowSize = (w + 1 + w) * (w + 1 + w);
			size_t number_cand0 = 0;
			for (size_t k = 0; k < (bi - ai + 1) * (bj - aj + 1); k++)
			{
				positionCand_orderDis_d[k + i_thread * nWindowSize] = 1;
				number_cand0++;
				for (size_t iband = 0; iband < nb; iband++)
				{
					size_t nGlbIdx = (k % (bi - ai + 1)) + ai + (k / (bi - ai + 1)) * ns + aj * ns + iband * ns * nl;
					size_t nGlbIdx_c = ci + ai + cj * ns + aj * ns + iband * ns * nl;
					if (abs(FineImg1_d[nGlbIdx] - FineImg1_d[nGlbIdx_c]) >= similar_th_d[nGlbIdx])
					{
						positionCand_orderDis_d[k + i_thread * nWindowSize] = 0;
						number_cand0--;
						break;
					}
				}
			}

			size_t order_dis_count = 0;
			for (size_t k = 0; k < (bi - ai + 1) * (bj - aj + 1); k++)
			{
				if (positionCand_orderDis_d[k + i_thread * nWindowSize] != 0)
				{
					double similar_cand_k = 0.0;
					for (size_t ib = 0; ib < nb; ib++)
					{
						size_t nGlbIdx = (k % (bi - ai + 1)) + ai + (k / (bi - ai + 1)) * ns + aj * ns + ib * ns * nl;
						size_t nGlbIdx_c = ci + ai + cj * ns + aj * ns + ib * ns * nl;
						similar_cand_k += abs(FineImg1_d[nGlbIdx] - FineImg1_d[nGlbIdx_c]) / (double)FineImg1_d[nGlbIdx_c];
					}
					mmap_order_dis_d[order_dis_count + i_thread * nWindowSize] = similar_cand_k;
					positionCand_orderDis_d[order_dis_count + i_thread * nWindowSize] = k;
					++order_dis_count;
				}
			}
			
			size_t number_cand = min(number_cand0, num_similar_pixel);

			sortSimilarPixel(mmap_order_dis_d + i_thread * nWindowSize, positionCand_orderDis_d + i_thread * nWindowSize, number_cand0, number_cand);

			if ((bi - ai + 1)*(bj - aj + 1) < (w*2.0 + 1)*(w*2.0 + 1))
			{
				for (size_t k = 0; k < number_cand; k++)
				{
					size_t nGlbIdx = positionCand_orderDis_d[k + i_thread * nWindowSize] + (nI + nJ * (w + 1)) * (w + 1 + w) * (w + 1 + w);
					D_D_cand_d[k + num_similar_pixel * i_thread] = sqrtf(double((ci - col_wind_d[nGlbIdx]) * (ci - col_wind_d[nGlbIdx]) + (cj - row_wind_d[nGlbIdx]) * (cj - row_wind_d[nGlbIdx]))) + 0.0000001;
				}
			}
			else
			{
				for (size_t k = 0; k < number_cand; k++)
					D_D_cand_d[k + num_similar_pixel * i_thread] = D_D_all_d[positionCand_orderDis_d[k + i_thread * nWindowSize]];
			}
			for (size_t k = 0; k < number_cand; k++)
			{
				double similar_cand_ind_same_class_k = 0.0;
				for (size_t ib = 0; ib < nb; ib++)
				{
					size_t nGlbIdx = (positionCand_orderDis_d[k + i_thread * nWindowSize] % (bi - ai + 1)) + ai + (positionCand_orderDis_d[k + i_thread * nWindowSize] / (bi - ai + 1)) * ns + aj * ns + ib * ns * nl;
					size_t nGlbIdx_c = ci + ai + cj * ns + aj * ns + ib * ns * nl;
					similar_cand_ind_same_class_k += abs(FineImg1_d[nGlbIdx] - FineImg1_d[nGlbIdx_c]) / (double)FineImg1_d[nGlbIdx_c];
				}
				D_D_cand_d[k + num_similar_pixel * i_thread] = (1.0 + D_D_cand_d[k + num_similar_pixel * i_thread] / w)*(similar_cand_ind_same_class_k + 1.0);
			}
			double temp_Total_C_D = 0.0;
			for (size_t k = 0; k < number_cand; k++)
			{
				temp_Total_C_D += 1.0 / D_D_cand_d[k + num_similar_pixel * i_thread];
			}
			//predict the value (formula 24)
			for (size_t iband = 0; iband < nb; iband++)
			{
				double temp_total_weight = 0;
				for (size_t k = 0; k < number_cand; k++)
				{
					size_t glbIdx = (positionCand_orderDis_d[k + i_thread * nWindowSize] % (bi - ai + 1)) + ai + (positionCand_orderDis_d[k + i_thread * nWindowSize] / (bi - ai + 1)) * ns + aj * ns + iband * ns * nl;
					temp_total_weight += (1.0 / D_D_cand_d[k + num_similar_pixel * i_thread] / temp_Total_C_D) * change_21_d[glbIdx];
				}
				//int _nIdxFine2 = i + j * ns + iband * ns * nl;
				size_t _nIdx = i + j * ns + iband * ns * nl;
				fine2_d[_nIdx] = FineImg1_d[_nIdx] + temp_total_weight;
				//revise the abnormal prediction

				if (fine2_d[_nIdx] <= DN_min || fine2_d[_nIdx] >= DN_max)
				{
					fine2_d[_nIdx] = FineImg1_d[_nIdx] + (CoarseImg2_d[_nIdx] - CoarseImg1_d[_nIdx]);
					if (fine2_d[_nIdx] > DN_max) fine2_d[_nIdx] = DN_max;
					if (fine2_d[_nIdx] < DN_min) fine2_d[_nIdx] = DN_min;
				}
			}
		}
	}
}

int cuFinalCalculation(parameter* p1, float* fine2, float *FineImg1, float* CoarseImg1, float* CoarseImg2, float *change_21, float *D_D_all, float* similar_th,
	size_t ns_block, size_t nl_block, size_t nb)
{
	size_t w = p1->w;
	dim3 dimGrid = p1->dimGrid;
	dim3 dimBlock = p1->dimBlock;
	size_t imgSize = ns_block * nl_block * nb;				//size of image, ns, nl, nb indicate the width, length, and the bands of img
	size_t nThread = dimGrid.x * dimBlock.x;				//size of 1 cycle
	size_t nWindowSize = (w + 1 + w) * (w + 1 + w);			//size of window, and w is the length of the searching window

	size_t *col_wind = new size_t[(w + 1) * (w + 1) * (w + 1 + w) * (w + 1 + w)]();
	size_t *row_wind = new size_t[(w + 1) * (w + 1) * (w + 1 + w) * (w + 1 + w)]();
	for (size_t i = 0; i < w + 1; i++)
	{
		for (size_t j = 0; j < w + 1; j++)
		{
			size_t ai, bi, aj, bj;
			if (i > w) ai = i - w; else ai = 0;
			bi = i + w;
			if (j > w) aj = j - w; else aj = 0;
			bj = j + w;

			for (size_t k = 0; k < bi - ai + 1; k++)
			{
				for (size_t l = 0; l < bj - aj + 1; l++)
				{
					col_wind[k + l * (bi - ai + 1) + (i + j * (w + 1)) * (w + 1 + w) * (w + 1 + w)] = k;
					row_wind[k + l * (bi - ai + 1) + (i + j * (w + 1)) * (w + 1 + w) * (w + 1 + w)] = l;
				}
			}
		}
	}

	float *CoarseImg1_d, *CoarseImg2_d;
	float *fine2_d;
	float *FineImg1_d, *change_21_d;
	float *D_D_all_d, *similar_th_d;
	size_t *col_wind_d, *row_wind_d;
	size_t *positionCand_orderDis_d;
	float *mmap_order_dis_d;
	double *D_D_cand_d;

	// Malloc the memory in device(GPU)
	hipMalloc((void**)&fine2_d, sizeof(float) * imgSize);
	hipMalloc((void**)&FineImg1_d, sizeof(float) * imgSize);
	hipMalloc((void**)&CoarseImg1_d, sizeof(float) * imgSize);
	hipMalloc((void**)&CoarseImg2_d, sizeof(float) * imgSize);
	hipMalloc((void**)&change_21_d, sizeof(float) * imgSize);
	hipMalloc((void**)&similar_th_d, sizeof(float) * imgSize);
	hipMalloc((void**)&D_D_all_d, sizeof(float) * nWindowSize);
	hipMalloc((void**)&col_wind_d, sizeof(size_t) * (w + 1) * (w + 1) * nWindowSize);
	hipMalloc((void**)&row_wind_d, sizeof(size_t) * (w + 1) * (w + 1) * nWindowSize);
	hipMalloc((void**)&positionCand_orderDis_d, sizeof(size_t) * nWindowSize * nThread);
	hipMalloc((void**)&mmap_order_dis_d, sizeof(float) * nWindowSize * nThread);
	hipMalloc((void**)&D_D_cand_d, sizeof(double) * p1->num_similar_pixel * nThread);


	// Copy data from host to device (CPU to GPU)
	hipMemcpy(fine2_d, fine2, sizeof(float) * imgSize, hipMemcpyHostToDevice);
	hipMemcpy(FineImg1_d, FineImg1, sizeof(float) * imgSize, hipMemcpyHostToDevice);
	hipMemcpy(CoarseImg1_d, CoarseImg1, sizeof(float) * imgSize, hipMemcpyHostToDevice);
	hipMemcpy(CoarseImg2_d, CoarseImg2, sizeof(float) * imgSize, hipMemcpyHostToDevice);
	hipMemcpy(change_21_d, change_21, sizeof(float) * imgSize, hipMemcpyHostToDevice);
	hipMemcpy(similar_th_d, similar_th, sizeof(float) * imgSize, hipMemcpyHostToDevice);
	hipMemcpy(D_D_all_d, D_D_all, sizeof(float) * nWindowSize, hipMemcpyHostToDevice);
	hipMemcpy(col_wind_d, col_wind, sizeof(size_t) * (w + 1) * (w + 1) * nWindowSize, hipMemcpyHostToDevice);
	hipMemcpy(row_wind_d, row_wind, sizeof(size_t) * (w + 1) * (w + 1) * nWindowSize, hipMemcpyHostToDevice);

	for (size_t iCycle = 0; iCycle * dimGrid.x * dimBlock.x < ns_block * nl_block; iCycle++)
	{
		size_t nCyl = (ns_block * nl_block / dimGrid.x / dimBlock.x + 20 - 1) / 20;
		if (nCyl != 0 && (iCycle % nCyl) == 0)
		{
			printf("\r*");
			for (size_t i = 0; i <= (iCycle / nCyl); i++)
				printf("-");
			for (size_t i = 0; i <= (ns_block * nl_block / (dimGrid.x * dimBlock.x) / nCyl) - (iCycle / nCyl); i++)
				printf(" ");
			printf("*");
		}
		cuFine2_1 << <dimGrid, dimBlock >> > (FineImg1_d, CoarseImg1_d, CoarseImg2_d, change_21_d, fine2_d, D_D_all_d, similar_th_d,
			col_wind_d, row_wind_d, positionCand_orderDis_d, mmap_order_dis_d, D_D_cand_d, ns_block, nl_block, nb, p1->background_band,
			p1->background, w, p1->DN_max, p1->DN_min, p1->num_similar_pixel, nThread, iCycle);

		CHECK(hipDeviceSynchronize());
	}


	// Copy result from device to host (GPU to CPU)
	hipMemcpy(fine2, fine2_d, sizeof(float) * imgSize, hipMemcpyDeviceToHost);

	hipFree(fine2_d);
	hipFree(FineImg1_d);
	hipFree(CoarseImg1_d);
	hipFree(CoarseImg2_d);
	hipFree(change_21_d);
	hipFree(D_D_all_d);
	hipFree(similar_th_d);
	hipFree(col_wind_d);
	hipFree(row_wind_d);
	hipFree(positionCand_orderDis_d);
	hipFree(mmap_order_dis_d);
	hipFree(D_D_cand_d);

	CHECK(hipDeviceReset());

	delete[]col_wind;
	delete[]row_wind;
	col_wind = NULL;
	row_wind = NULL;

	return 0;
}


